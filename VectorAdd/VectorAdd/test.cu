#include <iostream>
#include <hip/hip_runtime.h>


#include <stdio.h>
#include <math.h>
using namespace std;

__global__ void vecAddKernel(float *A, float *B, float *C, int n) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < n)
		C[i] = A[i] + B[i];
}

__host__ void vecAdd(float* A, float* B, float* C, int n) {
	int size = n*sizeof(float);
	float *d_A, *d_B, *d_C;

	//Allocate device(GPU) memory for A,B, C
	hipMalloc((void **) &d_A, size);
	hipMalloc((void **) &d_B, size);
	hipMalloc((void **) &d_C, size);

	//Copy A, B to device memory
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	//Kernel launch code to have the device to perform actual vector addition
	//cout << "Attempting to launch the kernel" << "\n";
	vecAddKernel <<< ceil(n / 256.0), 256 >>> (d_A, d_B, d_C, n);

	//Copy C from the device memory
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	//print values
	cout << ceil(*A) << "\n";
	cout << ceil(*B) << "\n";
	cout << ceil(*C) << "\n";
	system("pause");

	//Free device vectors
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

int main() {
	int N;
	cout << "Provide value for the length of the vector to be processed:";
	cin >> N;

	//Allocate input vectors h_A, h_b, h_C
	float* h_A = (float*)malloc(N * sizeof(float));
	float* h_B = (float*)malloc(N * sizeof(float));
	float* h_C = (float*)malloc(N * sizeof(float));
	vecAdd(h_A, h_B, h_C, N);
	free(h_A);
	free(h_B);
	free(h_C);
}

